#include "hip/hip_runtime.h"
﻿#include "stdafx.h"
#include "sunsky.cuh"

#include "Bbox.h"
#include "Rays.h"
#include "Scene.h"
#include "assert_cuda.h"
#include "cuda_surface_types.h"
#include ""
#include "surface_functions.h"
#include "variables.h"

surface<void, cudaSurfaceType2D> surf;
texture<float, hipTextureTypeCubemap> skybox;

__device__ unsigned int RandomInt(unsigned int& seed) {
  seed ^= seed << 13;
  seed ^= seed >> 17;
  seed ^= seed << 5;
  return seed;
}

__device__ float RandomFloat(unsigned int& seed) {
  return RandomInt(seed) * 2.3283064365387e-10f;
}

__device__ float RandomFloat2(unsigned int& seed) {
  return (RandomInt(seed) >> 16) / 65535.0f;
}

enum Refl_t { DIFF, SPEC, REFR };

inline __host__ __device__ float dot(const glm::vec4& v1, const glm::vec3& v2) {
  return v1.x * v2.x + v1.y * v2.y + v1.z * v2.z;
}

struct Sphere {
  float radius;
  glm::vec3 position, color;
  Refl_t refl;

  __device__ float intersect(const Ray& r) const {
    glm::vec3 op = position - r.orig;
    float t;
    float b = glm::dot(op, r.dir);
    float disc = b * b - dot(op, op) + radius * radius;
    if (disc < 0)
      return 0;
    else
      disc = sqrtf(disc);
    return (t = b - disc) > epsilon ? t : ((t = b + disc) > epsilon ? t : 0);
  }

  __device__ glm::vec3 random_point(unsigned int& seed) const {
    float u = RandomFloat(seed);
    float v = RandomFloat(seed);

    float cosPhi = 2.0f * u - 1.0f;
    float sinPhi = sqrt(1.0f - cosPhi * cosPhi);
    float theta = 2 * pi * v;

    float x = radius * sinPhi * sin(theta);
    float y = radius * cosPhi;
    float z = radius * sinPhi * cos(theta);

    return position + glm::vec3(x, y, z);
  }
};

__constant__ Sphere spheres[5];

__device__ inline bool intersect_scene(const Ray& ray, float& t, int& id,
                                       int& geometry_type,
                                       Scene::GPUScene sceneData) {
  float n = sizeof(spheres) / sizeof(Sphere), d;
  float inf = 1e20f;
  t = 1e20f;

  for (int i = int(n); i--;) {
    if ((d = spheres[i].intersect(ray)) && d < t) {
      t = d;
      id = i;
      geometry_type = 1;
    }
  }

  int triangle_id = -1;
  glm::vec3 pointHitInWorldSpace;
  float hit_distance = 1e20f;
 // for (int i = 0; i < 12; ++i) {
	//float dist =  sceneData.CUDACachedBVH.primitives[i].intersect(ray);
 //   if (dist > epsilon) {
 //         if (dist < hit_distance) {
	//		hit_distance = dist;
 //           triangle_id = i;
	//	  }
	//}
 // }
 // Triangle tri;
 // tri.vert = glm::vec3{0, 0, 0};
 // glm::vec3 vert2 = glm::vec3{0, 0, 4};
 // glm::vec3 vert3 = glm::vec3{4, 0, 0};
 // tri.e1 = vert2 - tri.vert;
 // tri.e2 = vert3 - tri.vert;
	//float dist =  tri.intersect(ray);
 //   if (dist > epsilon) {
 //         if (dist < hit_distance) {
	//		hit_distance = dist;
	//	  }
	//}

  //if (hit_distance < t) {
  //  t = hit_distance;
  //  geometry_type = 2;
  //  id = triangle_id;
  //}
    if (sceneData.CUDACachedBVH.intersect(ray, t, id)) {
      geometry_type = 2;
    }
  // if (sceneData.CUDACachedBVH.intersect(rayToIntersect,
  // lowestIntersectT,hitIndex)){
  //	geometryType = GeomType::Triangle;
  //}
  //// Intersection is too far, stop recursion
  // if (lowestIntersectT >= HugeEpsilon){
  //	totalColor += (rayColorMask *
  //computeBackgroundColor(rayToIntersect.direction)); 	return totalColor;
  //}

  // if (geometryType == GeomType::Triangle) {
  //	const Triangle &triangle = sceneData.CUDACachedBVH.primitives[hitIndex];
  //	//refltype  = ReflectiveType::Coat;
  //	refltype  =static_cast<ReflectiveType>(triangle.materialType);
  //	albedo = triangle.color;
  //	intersectionPoint = originInWorldSpace + rayInWorldSpace *
  //lowestIntersectT; 	normal = cross(triangle.e1,triangle.e2); 	normal =
  //normalize(normal); 	orientedNormal = dot(normal, rayInWorldSpace) < 0 ?
  //normal : normal * -1;
  //	//albedo    = make_float3(1.0,0.8,0.1);
  //	emittance = make_float3(0);
  //	//totalColor += (rayColorMask * emittance);
  //}
  // Intersect BVH
  // if (BVH_IntersectTriangles(cudaBVHindexesOrTrilists, ray, -1, triangle_id,
  //                           hit_distance, cudaBVHlimits,
  //                           cudaTriangleIntersectionData, cudaTriIdxList)) {
  //  if (hit_distance < t) {
  //    t = hit_distance;
  //    geometry_type = 2;
  //    id = triangle_id;
  //  }
  //}
  return t < inf;
}

__device__ glm::vec3 radiance(Ray& ray, unsigned int& seed,
                              Scene::GPUScene sceneData) {
  glm::vec3 color = {1.f, 1.f, 1.f};
  glm::vec3 direct = {0.f, 0.f, 0.f};

  int geometry_type = 0;
  int reflection_type;

  float distance;
  int id;
  for (int bounces = 0; bounces < 4; bounces++) {
    if (!intersect_scene(ray, distance, id, geometry_type, sceneData)) {
		return direct + color * (bounces > 0 ? sky(ray.dir) : sunsky(ray.dir));
    }

    glm::vec3 position = ray.orig + ray.dir * distance;
    glm::vec3 normal;
    switch (geometry_type) {
      case 1:
        const Sphere& object = spheres[id];
        normal = (position - object.position) / object.radius;
        color *= object.color;
        reflection_type = object.refl;
        break;
      case 2:
        //return {255, 0, 0};
        Triangle* triangle = &(sceneData.CUDACachedBVH.primitives[id]);
        normal = glm::cross(triangle->e1, triangle->e2);
        normal = glm::normalize(normal);
        // color *= glm::vec3(1, 1, 1);
        reflection_type = DIFF;
        break;
    }

    bool outside = dot(normal, ray.dir) < 0;
    normal =
        outside
            ? normal
            : normal * -1.f;  // make n front facing is we are inside an object
    ray.orig = position + normal * epsilon;

    switch (reflection_type) {
      case DIFF: {
        // Random direction in hemisphere
        float r1 = 2.f * pi * RandomFloat(seed);
        float r2 = RandomFloat(seed);
        float r2s = sqrt(r2);

        // Transform to hemisphere coordinate system
        const glm::vec3 u = (
            glm::cross((abs(normal.x) > .1f ? glm::vec3(0.f, 1.f, 0.f)
                                            : glm::vec3(1.f, 0.f, 0.f)),
                       normal));
        const glm::vec3 v = cross(normal, u);
        // Get sample on hemisphere
        const glm::vec3 d = (u * cos(r1) * r2s + v * sin(r1) * r2s +
                                      normal * sqrt(1 - r2));

        glm::vec3 sunSampleDir =
            getConeSample(sunDirection, 1.0f - sunAngularDiameterCos, seed);
        float sunLight = dot(normal, sunSampleDir);

        Ray shadow_ray = Ray(position + normal * 0.01f, sunSampleDir);
        float shadow_ray_distance;
        int shadow_ray_id;

        if (sunLight > 0.0 &&
            !intersect_scene(shadow_ray, shadow_ray_distance, shadow_ray_id,
                             geometry_type, sceneData)) {
          direct += color * sun(sunSampleDir) * sunLight * 1E-5f;
        }

        ray.dir = d;
        break;
      }
      case SPEC: {
        ray.dir = reflect(ray.dir, normal);
        break;
      }
      case REFR: {
        float n1 = outside ? 1.2f : 1.0f;
        float n2 = outside ? 1.0f : 1.2f;

        float r0 = (n1 - n2) / (n1 + n2);
        r0 *= r0;
        float fresnel =
            r0 + (1. - r0) * pow(1.0 - abs(dot(ray.dir, normal)), 5.);

        if (RandomFloat(seed) < fresnel) {
          ray.dir = reflect(ray.dir, normal);
        } else {
          ray.orig = position - normal * 2.f * epsilon;
          ray.dir = glm::refract(ray.dir, normal, n2 / n1);
        }
        break;
      }
    }
  }

  return direct;
}

__global__ void primary_rays(glm::vec3 camera_right, glm::vec3 camera_up,
                             glm::vec3 camera_direction, glm::vec3 O,
                             unsigned int frame, glm::vec4* blit_buffer,
                             Scene::GPUScene sceneData) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= render_width || y >= render_height) {
    return;
  }

  const float normalized_i = (x / (float)render_width) - 0.5f;
  const float normalized_j =
      ((render_height - y) / (float)render_height) - 0.5f;

  glm::vec3 direction =
      camera_direction + normalized_i * camera_right + normalized_j * camera_up;
  direction = normalize(direction);

  unsigned int seed = (frame * x * 147565741) * 720898027 * y;

  glm::vec3 r = radiance(Ray(O, direction), seed, sceneData);

  const int index = y * render_width + x;
  blit_buffer[index] += glm::vec4(r.x, r.y, r.z, 1);
}

__global__ void blit_onto_framebuffer(glm::vec4* blit_buffer, unsigned frames) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= render_width || y >= render_height) {
    return;
  }

  const int index = y * render_width + x;
  glm::vec4 color = blit_buffer[index] / (float)frames;
  surf2Dwrite<glm::vec4>(
      glm::pow(color / (color + 1.f), glm::vec4(1.0f / 2.2f)), surf,
      x * sizeof(glm::vec4), y, hipBoundaryModeZero);
}

bool first_time = true;
bool reset_buffer = false;
unsigned int frame = 0;
unsigned int hold_frame = 0;

hipError_t launch_kernels(hipArray_const_t array, glm::vec4* blit_buffer,
                           Scene::GPUScene sceneData) {
  if (first_time) {
    first_time = false;

    Sphere sphere_data[5] = {{16.5, {0, 40, 16.5f}, {1, 1, 1}, DIFF},
                             {16.5, {40, 0, 16.5f}, {1, 1, 1}, REFR},
                             {16.5, {-40, 0, 16.5f}, {1, 1, 1}, SPEC},
                             {1e4f, {0, 0, -1e4f - 20}, {1, 1, 1}, DIFF},
                             {40, {0, -80, 18.0f}, {1.0, 0.0, 0.0}, DIFF}};

    hipMemcpyToSymbol(HIP_SYMBOL(spheres), sphere_data, 5 * sizeof(Sphere));

    float sun_angular = cos(sunSize * pi / 180.0);
    cuda(MemcpyToSymbol(sunAngularDiameterCos, &sun_angular, sizeof(float)));
  }

  hipError_t cuda_err;
  static glm::vec3 last_pos;
  static glm::vec3 last_dir;

  cuda_err = cuda(BindSurfaceToArray(surf, array));

  if (cuda_err) {
    return cuda_err;
  }

  const glm::vec3 camera_right =
      glm::normalize(glm::cross(camera.direction, camera.up)) * 1.5f *
      ((float)render_width / render_height);
  const glm::vec3 camera_up =
      glm::normalize(glm::cross(camera_right, camera.direction)) * 1.5f;

  reset_buffer = last_pos != camera.position || last_dir != camera.direction;

  if (sun_position_changed) {
    sun_position_changed = false;
    reset_buffer = true;
    cuda(MemcpyToSymbol(SunPos, &sun_position, sizeof(glm::vec2)));
    glm::vec3 sun_direction = glm::normalize(fromSpherical(
        (sun_position - glm::vec2(0.0, 0.5)) * glm::vec2(6.28f, 3.14f)));
    cuda(MemcpyToSymbol(sunDirection, &sun_direction, sizeof(glm::vec3)));
  }

  if (reset_buffer) {
    reset_buffer = false;
    hipMemset(blit_buffer, 0, render_width * render_height * sizeof(float4));
    hold_frame = 1;
  }

  dim3 threads(8, 8, 1);
  dim3 blocks(render_width / threads.x, render_height / threads.y, 1);
  primary_rays<<<blocks, threads>>>(camera_right, camera_up, camera.direction,
                                    camera.position, frame, blit_buffer,
                                    sceneData);
  threads  =  dim3(16, 16, 1);
  blocks = dim3(render_width / threads.x, render_height / threads.y, 1);
  blit_onto_framebuffer<<<blocks, threads>>>(blit_buffer, hold_frame);

  frame++;
  hold_frame++;
  last_pos = camera.position;
  last_dir = camera.direction;

  return hipSuccess;
}