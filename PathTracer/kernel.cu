#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "sunsky.cuh"

#include "assert_cuda.h"
#include "cuda_surface_types.h"
#include ""
#include "surface_functions.h"

#include "cuda_definitions.h"

constexpr int NUM_SPHERES = 7;
constexpr float VERY_FAR = 1e20f;
constexpr int MAX_BOUNCES = 5;

surface<void, cudaSurfaceType2D> surf;
texture<float, hipTextureTypeCubemap> skybox;

//"Xorshift RNGs" by George Marsaglia
//http://excamera.com/sphinx/article-xorshift.html
__device__ unsigned int RandomInt(unsigned int& seed) {
	seed ^= seed << 13;
	seed ^= seed >> 17;
	seed ^= seed << 5;
	return seed;
}

//Random float between [0,1).
__device__ float RandomFloat(unsigned int& seed) {
	return RandomInt(seed) * 2.3283064365387e-10f;
}

__device__ float RandomFloat2(unsigned int& seed) {
	return (RandomInt(seed) >> 16) / 65535.0f;
}

enum Refl_t { DIFF,
			  SPEC,
			  REFR,
			  PHONG };

inline __host__ __device__ float dot(const glm::vec4& v1, const glm::vec3& v2) {
	return v1.x * v2.x + v1.y * v2.y + v1.z * v2.z;
}

struct Sphere {
	float radius;
	glm::vec3 position, color;
	glm::vec3 emmission;
	Refl_t refl;

	__device__ float intersect(const RayQueue& r) const {
		glm::vec3 op = position - r.origin;
		float t;
		float b = glm::dot(op, r.direction);
		float disc = b * b - dot(op, op) + radius * radius;
		if (disc < 0)
			return 0;

		disc = sqrtf(disc);
		return (t = b - disc) > epsilon ? t : ((t = b + disc) > epsilon ? t : 0);
	}

	__device__ float intersect_simple(const ShadowQueue& r) const {
		glm::vec3 op = position - r.origin;
		float t;
		float b = glm::dot(op, r.direction);
		float disc = b * b - dot(op, op) + radius * radius;
		if (disc < 0)
			return 0;

		disc = sqrtf(disc);
		return (t = b - disc) > epsilon ? t : ((t = b + disc) > epsilon ? t : 0);
	}

	__device__ glm::vec3 random_point(unsigned int& seed) const {
		float u = RandomFloat(seed);
		float v = RandomFloat(seed);

		float cosPhi = 2.0f * u - 1.0f;
		float sinPhi = sqrt(1.0f - cosPhi * cosPhi);
		float theta = 2 * pi * v;

		float x = radius * sinPhi * sin(theta);
		float y = radius * cosPhi;
		float z = radius * sinPhi * cos(theta);

		return position + glm::vec3(x, y, z);
	}
};

__constant__ Sphere spheres[NUM_SPHERES];

__device__ inline bool intersect_scene(RayQueue& ray, Scene::GPUScene sceneData) {
	float d;
	ray.distance = VERY_FAR;

	for (int i = NUM_SPHERES; i--;) {
		//d = spheres[i].intersect(ray);
		if ((d = spheres[i].intersect(ray)) && d < ray.distance) {
			ray.distance = d;
			ray.identifier = i;
			ray.geometry_type = GeometryType::Sphere;
		}
	}

	if (sceneData.CUDACachedBVH.intersect(ray)) {
		ray.geometry_type = GeometryType::Triangle;
	}
	return ray.distance < VERY_FAR;
}

__device__ inline bool intersect_scene_simple(ShadowQueue& ray, Scene::GPUScene sceneData) {
	for (int i = NUM_SPHERES; i--;) {
		float d = spheres[i].intersect_simple(ray);
		if (d != 0) {
			return true;
		}
	}

	return sceneData.CUDACachedBVH.intersectSimple(ray);
}

__device__ unsigned int shadow_ray_cnt = 0;
__device__ unsigned int primary_ray_cnt = 0;
__device__ unsigned int start_position = 0;
__device__ unsigned int raynr_shade = 0;
__device__ unsigned int raynr_primary = 0;
__device__ unsigned int raynr_extend = 0;

__global__ void zero_variables() {
	shadow_ray_cnt = 0;
	start_position += ray_queue_buffer_size - primary_ray_cnt;
	start_position = start_position % (render_width * render_height);
	primary_ray_cnt = 0;
	raynr_shade = 0;
	raynr_primary = 0;
	raynr_extend = 0;
}

/// Generate primary rays
__global__ void primary_rays(RayQueue* queue, glm::vec3 camera_right, glm::vec3 camera_up, glm::vec3 camera_direction, glm::vec3 O) {

	while (true) {
		unsigned int index = atomicAdd(&raynr_primary, 1);

		if (primary_ray_cnt + index > ray_queue_buffer_size - 1) {
			return;
		}

		const int x = ((start_position + index) % render_width);
		const int y = ((start_position + index) / render_width) % render_height;

		const float normalized_i = (x / (float)render_width) - 0.5f;
		const float normalized_j = ((render_height - y) / (float)render_height) - 0.5f;

		glm::vec3 direction = camera_direction + normalized_i * camera_right + normalized_j * camera_up;
		direction = normalize(direction);

		queue[primary_ray_cnt + index] = { O, direction, { 1, 1, 1 }, 0, 0, 0, x, y };
	}
}

/// Advance the ray segments once
__global__ void __launch_bounds__(128, 8) extend(RayQueue* queue, Scene::GPUScene sceneData) {

	while (true) {
		unsigned int index = atomicAdd(&raynr_extend, 1);

		if (index > ray_queue_buffer_size - 1) {
			return;
		}

		RayQueue& ray = queue[index];

		ray.distance = VERY_FAR;
		//sceneData.CUDACachedBVH.intersect(ray);
		intersect_scene(ray, sceneData);
	}
}

/// Process collisions and spawn extension and shadow rays
__global__ void __launch_bounds__(128, 8) shade(RayQueue* queue, RayQueue* queue2, ShadowQueue* shadowQueue, Scene::GPUScene sceneData, glm::vec4* blit_buffer, unsigned int frame) {

	while (true) {
		unsigned int index = atomicAdd(&raynr_shade, 1);

		if (index > ray_queue_buffer_size - 1) {
			return;
		}

		int new_frame = 0;
		RayQueue& ray = queue[index];
		glm::vec3 color = glm::vec3(0.f);

		unsigned int seed = (frame * ray.x * 147565741) * 720898027 * index;
		int reflection_type = DIFF;

		if (ray.distance < VERY_FAR) {
			ray.origin += ray.direction * ray.distance;
			ray.lastSpecular = false;

			glm::vec3 normal;
			if (ray.geometry_type == GeometryType::Sphere) {
				const Sphere& object = spheres[ray.identifier];
				normal = (ray.origin - object.position) / object.radius;
				reflection_type = object.refl;
				color = color + (ray.direct * object.emmission);
				ray.direct *= object.color;
			} else {
				Triangle* triangle = &(sceneData.CUDACachedBVH.primitives[ray.identifier]);
				normal = glm::normalize(glm::cross(triangle->e1, triangle->e2));
				reflection_type = DIFF;
			}

			bool outside = dot(normal, ray.direction) < 0;
			normal = outside ? normal : normal * -1.f; // make n front facing is we are inside an object

			ray.origin += normal * epsilon;

			switch (reflection_type) {
			case DIFF: {

				// Generate new shadow ray
				glm::vec3 sunSampleDir = getConeSample(sunDirection, 1.0f - sunAngularDiameterCos, seed);
				float sunLight = dot(normal, sunSampleDir);

				// < 0.f means sun is behind the surface
				if (sunLight > 0.f) {
					unsigned shadow_index = atomicAdd(&shadow_ray_cnt, 1);
					//shadowQueue[shadow_index] = { ray.origin, sunSampleDir, sunLight, ray.y * render_width + ray.x };
					ShadowQueue rayy = { ray.origin, sunSampleDir, sunLight, ray.y * render_width + ray.x };

					if (!intersect_scene_simple(rayy, sceneData)) {
						color += ray.direct * (sun(rayy.direction) * rayy.sunlight * 1E-5f);
					}
				}

				if (ray.bounces < MAX_BOUNCES) {
					float r1 = 2.f * pi * RandomFloat(seed);
					float r2 = RandomFloat(seed);
					float r2s = sqrt(r2);

					// Transform to hemisphere coordinate system
					const glm::vec3 u = glm::normalize(glm::cross((abs(normal.x) > .1f ? glm::vec3(0.f, 1.f, 0.f) : glm::vec3(1.f, 0.f, 0.f)), normal));
					const glm::vec3 v = cross(normal, u);
					// Get sample on hemisphere
					const glm::vec3 d = glm::normalize(u * cos(r1) * r2s + v * sin(r1) * r2s + normal * sqrt(1 - r2));
					ray.direction = d;
				}

				break;
			}
			case SPEC: {
				ray.lastSpecular = true;
				ray.direction = reflect(ray.direction, normal);
				break;
			}
			case REFR: {

				float n1 = outside ? 1.2f : 1.0f;
				float n2 = outside ? 1.0f : 1.2f;

				float r0 = (n1 - n2) / (n1 + n2);
				r0 *= r0;
				float fresnel = r0 + (1. - r0) * pow(1.0 - abs(dot(ray.direction, normal)), 5.);

				if (RandomFloat(seed) < fresnel) {
					ray.direction = reflect(ray.direction, normal);
				} else {
					ray.origin = ray.origin - normal * 2.f * epsilon;
					ray.direction = glm::refract(ray.direction, normal, n2 / n1);
				}
				break;
			}
			case PHONG: {
				// compute random perturbation of ideal reflection vector
				// the higher the phong exponent, the closer the perturbed vector
				// is to the ideal reflection direction
				float phi = 2 * pi * RandomFloat(seed);
				float r2 = RandomFloat(seed);
				float phongexponent = 25;
				float cosTheta = powf(1 - r2, 1.0f / (phongexponent + 1));
				float sinTheta = sqrtf(1 - cosTheta * cosTheta);

				/* 
				Create orthonormal basis uvw around reflection vector with 
				hitpoint as origin w is ray direction for ideal reflection
			 */
				glm::vec3 w;
				w = ray.direction - normal * 2.0f * dot(normal, ray.direction);
				w = normalize(w);

				// Transform to hemisphere coordinate system
				const glm::vec3 u = glm::normalize(
					glm::cross((abs(normal.x) > .9f ? glm::vec3(0.f, 1.f, 0.f)
													: glm::vec3(1.f, 0.f, 0.f)),
							   normal));
				const glm::vec3 v = cross(w, u);
				// Get sample on hemisphere
				// compute cosine weighted random ray direction on hemisphere

				glm::vec3 d = u * cosf(phi) * sinTheta + v * sinf(phi) * sinTheta + w * cosTheta;
				d = normalize(d);

				glm::vec3 sunSampleDir = getConeSample(sunDirection, 1.0f - sunAngularDiameterCos, seed);
				float sunLight = dot(normal, sunSampleDir);

				//SunLight is cos of sampleDir to normal. For phong we weight it proportional to cos(theta) ^ phongExponent
				sunLight = powf(sunLight, phongexponent);
				if (sunLight > 0.f) {
					unsigned shadow_index = atomicAdd(&shadow_ray_cnt, 1);
					//shadowQueue[shadow_index] = { ray.origin, sunSampleDir, sunLight, ray.y * render_width + ray.x };
					ShadowQueue rayy = { ray.origin, sunSampleDir, sunLight, ray.y * render_width + ray.x };

					if (!intersect_scene_simple(rayy, sceneData)) {
						color += ray.direct * (sun(rayy.direction) * rayy.sunlight * 1E-5f);
					}
				}

				ray.origin = ray.origin + w * epsilon; // scene size dependent
				ray.direction = d;

				break;
			}
			}

			if (ray.bounces < MAX_BOUNCES) {
				ray.bounces++;

				unsigned primary_index = atomicAdd(&primary_ray_cnt, 1);
				queue2[primary_index] = ray;
			} else {

				new_frame++;
			}

		} else {
			// Don't generate new extended ray
			color += (ray.lastSpecular == false) ? ray.direct * sky(ray.direction) : ray.direct * sunsky(ray.direction);
			new_frame++;
		}

		atomicAdd(&blit_buffer[ray.y * render_width + ray.x].r, color.r);
		atomicAdd(&blit_buffer[ray.y * render_width + ray.x].g, color.g);
		atomicAdd(&blit_buffer[ray.y * render_width + ray.x].b, color.b);
		atomicAdd(&blit_buffer[ray.y * render_width + ray.x].a, new_frame);
	}
}

/// Proccess shadow rays
__global__ void connect(ShadowQueue* queue, Scene::GPUScene sceneData, glm::vec4* blit_buffer) {
	//const int index = blockIdx.x * blockDim.x + threadIdx.x;

	//if (index > shadow_ray_count - 1) {
	//	return;
	//}

	//ShadowQueue& ray = queue[index];

	//if (!sceneData.CUDACachedBVH.intersectSimple(ray)) {
	//	glm::vec3 color = sun(ray.direction) * ray.sunlight * 1E-5f;
	//primary_queue[ray.primary_index].direct += color;
	//	blit_buffer[ray.buffer_index] += glm::vec4(color, 0);
	//}
}

__global__ void blit_onto_framebuffer(glm::vec4* blit_buffer) {
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= render_width || y >= render_height) {
		return;
	}

	const int index = y * render_width + x;
	glm::vec4 color = blit_buffer[index];
	glm::vec4 cl = glm::vec4(color.r, color.g, color.b, 1) / color.a;
	cl.a = 1;

	surf2Dwrite<glm::vec4>(glm::pow(cl / (cl + 1.f), glm::vec4(1.0f / 2.2f)), surf, x * sizeof(glm::vec4), y, hipBoundaryModeZero);
}

hipError_t launch_kernels(hipArray_const_t array, glm::vec4* blit_buffer, Scene::GPUScene sceneData, RayQueue* queue, RayQueue* queue2, ShadowQueue* shadow_queue) {
	static bool first_time = true;
	static bool reset_buffer = false;
	static unsigned int frame = 0;

	if (first_time) {
		first_time = false;

		Sphere sphere_data[NUM_SPHERES] = { { 16.5, { 0, 40, 16.5f }, { 1, 1, 1 }, { 0, 0, 0 }, DIFF },
											{ 16.5, { 40, 0, 16.5f }, { 1, 1, 1 }, { 0, 0, 0 }, REFR },
											{ 16.5, { -40, 0, 16.5f }, { 0.6, 0.5, 0.4 }, { 0, 0, 0 }, PHONG },
											{ 16.5, { -40, -50, 16.5f }, { 0.6, 0.5, 0.4 }, { 0, 0, 0 }, SPEC },
											{ 1e4f, { 0, 0, -1e4f - 20 }, { 1, 1, 1 }, { 0, 0, 0 }, PHONG },
											{ 20, { 0, -80, 20 }, { 1.0, 0.0, 0.0 }, { 0, 0, 0 }, DIFF },
											{ 30, { 0, -80, 120.0f }, { 0.0, 1.0, 0.0 }, { 2, 2, 2 }, DIFF } };
		hipMemcpyToSymbol(HIP_SYMBOL(spheres), sphere_data, NUM_SPHERES * sizeof(Sphere));

		float sun_angular = cos(sunSize * pi / 180.f);
		cuda(MemcpyToSymbol(sunAngularDiameterCos, &sun_angular, sizeof(float)));
	}

	hipError_t cuda_err;
	static glm::vec3 last_pos;
	static glm::vec3 last_dir;

	cuda_err = cuda(BindSurfaceToArray(surf, array));

	if (cuda_err) {
		return cuda_err;
	}

	const glm::vec3 camera_right = glm::normalize(glm::cross(camera.direction, camera.up)) * 1.5f * ((float)render_width / render_height);
	const glm::vec3 camera_up = glm::normalize(glm::cross(camera_right, camera.direction)) * 1.5f;

	reset_buffer = last_pos != camera.position || last_dir != camera.direction;

	if (sun_position_changed) {
		sun_position_changed = false;
		reset_buffer = true;
		cuda(MemcpyToSymbol(SunPos, &sun_position, sizeof(glm::vec2)));
		glm::vec3 sun_direction = glm::normalize(fromSpherical((sun_position - glm::vec2(0.0, 0.5)) * glm::vec2(6.28f, 3.14f)));
		cuda(MemcpyToSymbol(sunDirection, &sun_direction, sizeof(glm::vec3)));
	}

	if (reset_buffer) {
		reset_buffer = false;
		hipMemset(blit_buffer, 0, render_width * render_height * sizeof(float4));

		int new_value = 0;
		cuda(MemcpyToSymbol(primary_ray_cnt, &new_value, sizeof(int)));
	}

	primary_rays<<<40, 128>>>(queue, camera_right, camera_up, camera.direction, camera.position);
	zero_variables<<<1, 1>>>();
	extend<<<40, 128>>>(queue, sceneData);
	shade<<<40, 128>>>(queue, queue2, shadow_queue, sceneData, blit_buffer, frame);
	//connect<<<40, 128>>>(shadow_queue, sceneData, blit_buffer);

	dim3 threads = dim3(16, 16, 1);
	dim3 blocks = dim3(render_width / threads.x, render_height / threads.y, 1);
	blit_onto_framebuffer<<<blocks, threads>>>(blit_buffer);

	cuda(DeviceSynchronize());

	frame++;
	//hold_frame++;
	last_pos = camera.position;
	last_dir = camera.direction;

	return hipSuccess;
}